#include "hip/hip_runtime.h"
/*
 * sha256.cu Implementation of SHA256 Hashing
 *
 * Date: 12 June 2019
 * Revision: 1
 * *
 * Based on the public domain Reference Implementation in C, by
 * Brad Conte, original code here:
 *
 * https://github.com/B-Con/crypto-algorithms
 *
 * This file is released into the Public Domain.
 */

 // SHA256 CUDA retrieved from https://github.com/mochimodev/cuda-hashing-algos/blob/master/

 /*************************** HEADER FILES ***************************/
#include <hip/hip_runtime.h>
#include ""
#include <stdlib.h>
#include <memory.h>
#include <time.h>
#include <limits.h>
#include <stdio.h>
#include <time.h>
extern "C" {
#include "sha256.cuh"
}


/****************************** DATA STRUCTURE ******************************/
typedef struct {
	unsigned char data[64];
	unsigned int datalen;
	unsigned long long bitlen;
	unsigned int state[8];
} CUDA_SHA256_CTX;


/****************************** MACROS ******************************/
#define SHA256_HASH_SIZE 32            // SHA256 outputs a 32 byte digest
#define SHA256_BLOCK_SIZE 64
#define ID_SIZE 11
#define NUM_BLOCKS 8192
#define NUM_THREADS 256

// Error checking from https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define CHECK_ERROR(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

/****************************** MACROS ******************************/
#ifndef ROTLEFT
#define ROTLEFT(a,b) (((a) << (b)) | ((a) >> (32-(b))))
#endif

#define ROTRIGHT(a,b) (((a) >> (b)) | ((a) << (32-(b))))

#define CH(x,y,z) (((x) & (y)) ^ (~(x) & (z)))
#define MAJ(x,y,z) (((x) & (y)) ^ ((x) & (z)) ^ ((y) & (z)))
#define EP0(x) (ROTRIGHT(x,2) ^ ROTRIGHT(x,13) ^ ROTRIGHT(x,22))
#define EP1(x) (ROTRIGHT(x,6) ^ ROTRIGHT(x,11) ^ ROTRIGHT(x,25))
#define SIG0(x) (ROTRIGHT(x,7) ^ ROTRIGHT(x,18) ^ ((x) >> 3))
#define SIG1(x) (ROTRIGHT(x,17) ^ ROTRIGHT(x,19) ^ ((x) >> 10))

/**************************** VARIABLES *****************************/
__constant__ unsigned int k[64] = {
	0x428a2f98,0x71374491,0xb5c0fbcf,0xe9b5dba5,0x3956c25b,0x59f111f1,0x923f82a4,0xab1c5ed5,
	0xd807aa98,0x12835b01,0x243185be,0x550c7dc3,0x72be5d74,0x80deb1fe,0x9bdc06a7,0xc19bf174,
	0xe49b69c1,0xefbe4786,0x0fc19dc6,0x240ca1cc,0x2de92c6f,0x4a7484aa,0x5cb0a9dc,0x76f988da,
	0x983e5152,0xa831c66d,0xb00327c8,0xbf597fc7,0xc6e00bf3,0xd5a79147,0x06ca6351,0x14292967,
	0x27b70a85,0x2e1b2138,0x4d2c6dfc,0x53380d13,0x650a7354,0x766a0abb,0x81c2c92e,0x92722c85,
	0xa2bfe8a1,0xa81a664b,0xc24b8b70,0xc76c51a3,0xd192e819,0xd6990624,0xf40e3585,0x106aa070,
	0x19a4c116,0x1e376c08,0x2748774c,0x34b0bcb5,0x391c0cb3,0x4ed8aa4a,0x5b9cca4f,0x682e6ff3,
	0x748f82ee,0x78a5636f,0x84c87814,0x8cc70208,0x90befffa,0xa4506ceb,0xbef9a3f7,0xc67178f2
};

unsigned int k_host[64] = {
	0x428a2f98,0x71374491,0xb5c0fbcf,0xe9b5dba5,0x3956c25b,0x59f111f1,0x923f82a4,0xab1c5ed5,
	0xd807aa98,0x12835b01,0x243185be,0x550c7dc3,0x72be5d74,0x80deb1fe,0x9bdc06a7,0xc19bf174,
	0xe49b69c1,0xefbe4786,0x0fc19dc6,0x240ca1cc,0x2de92c6f,0x4a7484aa,0x5cb0a9dc,0x76f988da,
	0x983e5152,0xa831c66d,0xb00327c8,0xbf597fc7,0xc6e00bf3,0xd5a79147,0x06ca6351,0x14292967,
	0x27b70a85,0x2e1b2138,0x4d2c6dfc,0x53380d13,0x650a7354,0x766a0abb,0x81c2c92e,0x92722c85,
	0xa2bfe8a1,0xa81a664b,0xc24b8b70,0xc76c51a3,0xd192e819,0xd6990624,0xf40e3585,0x106aa070,
	0x19a4c116,0x1e376c08,0x2748774c,0x34b0bcb5,0x391c0cb3,0x4ed8aa4a,0x5b9cca4f,0x682e6ff3,
	0x748f82ee,0x78a5636f,0x84c87814,0x8cc70208,0x90befffa,0xa4506ceb,0xbef9a3f7,0xc67178f2
};

/*********************** FUNCTION DEFINITIONS ***********************/
__host__ __device__ __forceinline__ void cuda_sha256_transform(CUDA_SHA256_CTX* ctx, const unsigned char data[])
{
	unsigned int a, b, c, d, e, f, g, h, i, j, t1, t2, m[64];

	for (i = 0, j = 0; i < 16; ++i, j += 4)
		m[i] = (data[j] << 24) | (data[j + 1] << 16) | (data[j + 2] << 8) | (data[j + 3]);
	for (; i < 64; ++i)
		m[i] = SIG1(m[i - 2]) + m[i - 7] + SIG0(m[i - 15]) + m[i - 16];

	a = ctx->state[0];
	b = ctx->state[1];
	c = ctx->state[2];
	d = ctx->state[3];
	e = ctx->state[4];
	f = ctx->state[5];
	g = ctx->state[6];
	h = ctx->state[7];

	for (i = 0; i < 64; ++i) {
#ifdef __CUDA_ARCH__
		t1 = h + EP1(e) + CH(e, f, g) + k[i] + m[i];
#else
		t1 = h + EP1(e) + CH(e, f, g) + k_host[i] + m[i];
#endif

		t2 = EP0(a) + MAJ(a, b, c);
		h = g;
		g = f;
		f = e;
		e = d + t1;
		d = c;
		c = b;
		b = a;
		a = t1 + t2;
	}

	ctx->state[0] += a;
	ctx->state[1] += b;
	ctx->state[2] += c;
	ctx->state[3] += d;
	ctx->state[4] += e;
	ctx->state[5] += f;
	ctx->state[6] += g;
	ctx->state[7] += h;
}

__host__ __device__ void cuda_sha256_init(CUDA_SHA256_CTX* ctx)
{
	ctx->datalen = 0;
	ctx->bitlen = 0;
	ctx->state[0] = 0x6a09e667;
	ctx->state[1] = 0xbb67ae85;
	ctx->state[2] = 0x3c6ef372;
	ctx->state[3] = 0xa54ff53a;
	ctx->state[4] = 0x510e527f;
	ctx->state[5] = 0x9b05688c;
	ctx->state[6] = 0x1f83d9ab;
	ctx->state[7] = 0x5be0cd19;
}

__host__ __device__ void cuda_sha256_update(CUDA_SHA256_CTX* ctx, const unsigned char data[], size_t len)
{
	unsigned int i;

	for (i = 0; i < len; ++i) {
		ctx->data[ctx->datalen] = data[i];
		ctx->datalen++;
		if (ctx->datalen == 64) {
			cuda_sha256_transform(ctx, ctx->data);
			ctx->bitlen += 512;
			ctx->datalen = 0;
		}
	}
}

__host__ __device__ void cuda_sha256_final(CUDA_SHA256_CTX* ctx, unsigned char hash[])
{
	unsigned int i;

	i = ctx->datalen;

	// Pad whatever data is left in the buffer.
	if (ctx->datalen < 56) {
		ctx->data[i++] = 0x80;
		while (i < 56)
			ctx->data[i++] = 0x00;
	}
	else {
		ctx->data[i++] = 0x80;
		while (i < 64)
			ctx->data[i++] = 0x00;
		cuda_sha256_transform(ctx, ctx->data);
		memset(ctx->data, 0, 56);
	}

	// Append to the padding the total message's length in bits and transform.
	ctx->bitlen += ctx->datalen * 8;
	ctx->data[63] = (unsigned char) (ctx->bitlen);
	ctx->data[62] = (unsigned char) (ctx->bitlen >> 8);
	ctx->data[61] = (unsigned char) (ctx->bitlen >> 16);
	ctx->data[60] = (unsigned char) (ctx->bitlen >> 24);
	ctx->data[59] = (unsigned char) (ctx->bitlen >> 32);
	ctx->data[58] = (unsigned char) (ctx->bitlen >> 40);
	ctx->data[57] = (unsigned char) (ctx->bitlen >> 48);
	ctx->data[56] = (unsigned char) (ctx->bitlen >> 56);
	cuda_sha256_transform(ctx, ctx->data);

	// Since this implementation uses little endian byte ordering and SHA uses big endian,
	// reverse all the bytes when copying the final state to the output hash.
	for (i = 0; i < 4; ++i) {
		hash[i] = (ctx->state[0] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 4] = (ctx->state[1] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 8] = (ctx->state[2] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 12] = (ctx->state[3] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 16] = (ctx->state[4] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 20] = (ctx->state[5] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 24] = (ctx->state[6] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 28] = (ctx->state[7] >> (24 - i * 8)) & 0x000000ff;
	}
}

__global__ void kernel_sha256_hash(unsigned char* indata, unsigned int inlen, unsigned char* outdata, unsigned int n_batch)
{
	unsigned int thread = blockIdx.x * blockDim.x + threadIdx.x;
	if (thread >= n_batch)
	{
		return;
	}
	unsigned char* in = indata + thread * inlen;
	unsigned char* out = outdata + thread * SHA256_HASH_SIZE;
	CUDA_SHA256_CTX ctx;
	cuda_sha256_init(&ctx);
	cuda_sha256_update(&ctx, in, inlen);
	cuda_sha256_final(&ctx, out);
}

__host__ __device__ unsigned char verify_zeroes(unsigned char* hash) {
	unsigned char num_zeroes = 0;
	unsigned char leading = 1;
	unsigned char i;

	for (i = 0; i < SHA256_HASH_SIZE; i++) {
		leading = leading && ((hash[i] | 0x0f) == 0x0f);
		num_zeroes += leading ? 1 : 0;
		leading = leading && ((hash[i] | 0xf0) == 0xf0);
		num_zeroes += leading ? 1 : 0;
	}
	return num_zeroes;
}

__global__ void kernel_mine_coin_child(CUDA_SHA256_CTX* hash_start, unsigned char* id, long long* result, long long start, unsigned char difficulty) {
	long long data_num = (blockIdx.x * blockDim.x + threadIdx.x) + start;
	unsigned char hash[SHA256_HASH_SIZE];

	CUDA_SHA256_CTX ctx;
	memcpy(&ctx, hash_start, sizeof(CUDA_SHA256_CTX));

	cuda_sha256_update(&ctx, (unsigned char*) &data_num, sizeof(data_num));
	cuda_sha256_update(&ctx, id, ID_SIZE);
	cuda_sha256_final(&ctx, hash);
	
	unsigned char num_zeroes = verify_zeroes(hash);
	if (num_zeroes < difficulty)
		return;
	*result = data_num;
}

extern "C" {
	long long cuda_mine_coin(const unsigned char* hash_start, const unsigned char* id, size_t hash_start_size, size_t id_size, unsigned char difficulty, int* latest_timestamp) {
		unsigned char* cuda_id;
		CUDA_SHA256_CTX* cuda_sha256_ctx;

		long long* cuda_result;
		long long host_result = -1;
		long long ret = -1;
		int timestamp = *latest_timestamp;

		CUDA_SHA256_CTX ctx;
		cuda_sha256_init(&ctx);
		cuda_sha256_update(&ctx, hash_start, hash_start_size);

		CHECK_ERROR(hipHostAlloc((void**)&cuda_result, sizeof(long long), hipHostMallocDefault));
		CHECK_ERROR(hipMalloc((void**)&cuda_sha256_ctx, sizeof(CUDA_SHA256_CTX)));
		CHECK_ERROR(hipMalloc((void**)&cuda_id, id_size));

		CHECK_ERROR(hipMemcpy(cuda_sha256_ctx, &ctx, sizeof(CUDA_SHA256_CTX), hipMemcpyHostToDevice));
		CHECK_ERROR(hipMemcpy(cuda_id, id, id_size, hipMemcpyHostToDevice));

		for (long long i = 0; i < LLONG_MAX - NUM_BLOCKS * NUM_THREADS; i += NUM_BLOCKS * NUM_THREADS) {
			host_result = -1;

			// Generate random start num for kernel
			long long start_num = rand();
			start_num = start_num << 31;
			start_num |= rand();
			// Prevent overflow when iterating in kernel
			start_num -= NUM_BLOCKS * NUM_THREADS;

			// Run kernel to compute hashes
			CHECK_ERROR(hipMemcpy(cuda_result, &host_result, sizeof(long long), hipMemcpyHostToDevice));
			kernel_mine_coin_child <<<NUM_BLOCKS, NUM_THREADS>>> (cuda_sha256_ctx, cuda_id, cuda_result, start_num, difficulty);
			CHECK_ERROR(hipMemcpy(&host_result, cuda_result, sizeof(long long), hipMemcpyDeviceToHost));

			// Check if valid hash was found
			if (host_result != -1) {
				ret = host_result;
				goto end;
			}

			// Check if coin is stale;
			if (*latest_timestamp > timestamp) {
				printf("Coin is stale. Ending iteration\n");
				ret = -1;
				goto end;
			}

		}

	end:
		CHECK_ERROR(hipFree(cuda_id));
		CHECK_ERROR(hipHostFree(cuda_result));
		CHECK_ERROR(hipFree(cuda_sha256_ctx));

		return ret;
	}
}